#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <life.h>
#include <lcutil.h>
#include <timestamp.h>

#include <utility>
#include <vector>

//#define DEBUG

#define SHARED

__global__ void nextGen(char* d_life, char* d_life_copy, const int size, int nblocks, dim3 dimBl) {
#ifndef SHARED

    int neighbors = 0;
    int first_in_row, down, up, left, right, upright, upleft, downright, downleft;

    int rows = dimBl.x + 2;
    int columns = dimBl.y + 2;

    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x;
    for (int cell = blockIdx.x * blockDim.x + threadIdx.x; cell < rows * columns; cell += blockDim.x * gridDim.x) {
            first_in_row = cell -  cell % columns;
            down         = (cell + columns) % (rows * columns);
            up           = (cell + rows * columns - columns) % (rows * columns);
            left         = (cell + rows * columns - 1) % columns + first_in_row;
            right        = (cell + rows * columns + 1) % columns + first_in_row;
            upleft       = (left + rows * columns - columns) % (rows * columns);
            downleft     = (left + rows * columns + columns) % (rows * columns);
            upright      = (right + rows * columns - columns) % (rows * columns);
            downright    = (right + rows * columns + columns) % (rows * columns);
#ifdef DEBUG       
            if (cell == 0 || cell ==4 || cell == 20 || cell == 24 || cell == 12)
                printf("I am cell %d and my neighbors are up %d down %d right %d left %d upright %d upleft %d downright %d downleft %d\n", cell, up, down, right, left, upright, upleft, downright, downleft);
#endif
            neighbors = *(d_life + upleft)   + *(d_life + up)   + *(d_life + upright) +
                        *(d_life + left)                +         *(d_life + right)   +
                        *(d_life + downleft) + *(d_life + down) + *(d_life + downright);
        
            if (neighbors == 3 || (neighbors == 2 && *(d_life_copy + cell) == 1))
                *(d_life_copy + cell) = 1;
            else
                *(d_life_copy + cell) = 0;
    }
#else
    int lrows = dimBl.y;
    int lcolumns = dimBl.x;
    __shared__ char* local = (char*)malloc(lrows * lcolumns * sizeof(char));

    for (int i = 0; i < lrows; i++) {
        for (int j = 0; j < lcolumns; j++) {

        }
    }
#endif
}

//////////////////////////////////////////////////////////////////////////////////////
// Plays the Game Of Life. It checks the contents of d_life,
// calculates the results, and stores them in d_life_copy. The living organisms
// are represented by a 1, and the dead organisms by a 0.
//////////////////////////////////////////////////////////////////////////////////////
extern "C" float GameOfLife(const int size, char* h_life, char* h_life_copy, int nblocks, dim3 dimBl, int generations) {
    // The grids that will be copied to the GPU
    char* d_life;
    char* d_life_copy;
    hipError_t err;

    err = hipMalloc((void**)&d_life, size * size * sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Could not allocate CUDA memory, with error code %d\n", err);
        return err;
    }

    err = hipMalloc((void**)&d_life_copy, size * size * sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Could not allocate CUDA memory, with error code %d\n", err);
        return err;
    }

    err = hipMemcpy(d_life, h_life, sizeof(char) * size * size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Could not copy to GPU memory, with error code %d\n", err);
        return err;
    }

    err = hipMemcpy(d_life_copy, h_life_copy, sizeof(char) * size * size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Could not copy to GPU memory, with error code %d\n", err);
        return err;
    }

    timestamp t_start;
    t_start = getTimestamp();

    for (int gen = 0; gen < generations; gen++) {
        nextGen <<<nblocks, dimBl>>> (d_life, d_life_copy, size, nblocks, dimBl);

#ifdef DEBUG
        hipMemcpy(h_life, d_life, sizeof(char) * size * size, hipMemcpyDeviceToHost);
        hipMemcpy(h_life_copy, d_life_copy, sizeof(char) * size * size, hipMemcpyDeviceToHost);
        printf("Generation %d\n", gen);
        printf("life\n");
        Print_grid(size, h_life);
        printf("life_copy\n");
        Print_grid(size, h_life_copy);
#endif
        /////////////////////////////////////////////////////////////////////////////////////////////////
        // Swap the addresses of the two tables. That way, we avoid copying the contents
        // of d_life to d_life_copy. Each round, the addresses are exchanged, saving time from running
        // a loop to copy the contents.
        /////////////////////////////////////////////////////////////////////////////////////////////////
        std::swap(d_life, d_life_copy);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Error synchronizing devices: %s\n", err);
        return err;
    }

    float msecs = getElapsedtime(t_start);

    err = hipFree(d_life);
    if (err != hipSuccess) {
        fprintf(stderr, "Error freeing GPU memory: %s\n", err);
        return err;
    }

    err = hipFree(d_life_copy);
    if (err != hipSuccess) {
        fprintf(stderr, "Error freeing GPU memory: %s\n", err);
        return err;
    }

    return msecs;
}